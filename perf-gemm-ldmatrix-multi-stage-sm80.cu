#include "hip/hip_runtime.h"
﻿#include <random>

#include "hip/hip_runtime.h"
#include ""
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "mma.h"
#include "hipblas.h"

#include "neo/layout.hpp"
#include "neo/tensor.hpp"
#include "neo/swizzle.hpp"
#include "neo/numeric/integral_constant.hpp"


template <typename T>
__global__ void gpu_compare_kernel(const T* x, const T* y, int n,
    float threshold, int* count,
    float* max_error) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= n) {
        return;
    }

    float v0 = x[idx];
    float v1 = y[idx];

    float diff = fabs(v0 - v1);
    if (diff > threshold) {
        atomicAdd(count, 1);

        // for positive floating point, there int representation is in the same
        // order.
        int int_diff = *((int*)(&diff));
        atomicMax((int*)max_error, int_diff);
    }
}

template <typename T>
void gpu_compare(const T* x, const T* y, int n, float threshold = 1.E-1) {
    int* num_count;
    float* max_error;
    hipMalloc(&num_count, sizeof(int));
    hipMalloc(&max_error, sizeof(float));
    hipMemset(num_count, 0, sizeof(int));
    hipMemset(max_error, 0, sizeof(float));

    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);
    gpu_compare_kernel << <grid, block >> > (x, y, n, threshold, num_count, max_error);
    int num = 0;
    float error = 0;
    hipMemcpy(&num, num_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&error, max_error, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (num == 0) {
        std::cout << "check ok, max_error = " << error << std::endl;
    }
    else {
        std::cout << "===============================" << std::endl <<
            "check fail: diff " << num << " max_error = " << error << std::endl <<
            "===============================" << std::endl;
    }
}

struct NeoConfig {
    using T = half;

    static constexpr int TILE_M = 128, TILE_N = 128, TILE_K = 32;
    static constexpr int MMA_M = 16, MMA_N = 16, MMA_K = 16;
    static constexpr int CopyUnitSize = 16;
    static constexpr int Threads = 128;
    static constexpr int WrapSize = 32;
    static constexpr int WrapCount = Threads / WrapSize;
    static constexpr int CopyCount = CopyUnitSize / sizeof(T);
    static constexpr int CopyColAB = TILE_K / CopyCount;
    static constexpr int CopyRowAB = Threads / CopyColAB;
    static constexpr int CopyColC = TILE_N / CopyCount;
    static constexpr int CopyRowC = Threads / CopyColC;
    static constexpr int Stage = 3;
    static constexpr int ShmAsize = TILE_M * TILE_K;
    static constexpr int ShmBsize = TILE_N * TILE_K;
    static constexpr int ShmCsize = TILE_M * TILE_N;
    static constexpr int ShmSize = std::max((ShmAsize + ShmBsize) * Stage, ShmCsize) * sizeof(T);

    using CopyTiledShapeA = decltype(neo::make_shape(neo::Int<CopyRowAB>{}, neo::Int<CopyColAB* CopyCount>{}));
    using CopyThrdShapeA = decltype(neo::make_shape(neo::Int<CopyRowAB>{}, neo::Int<CopyColAB>{}));
    using SwizzleA = decltype(neo::Swizzle<3, 3, 3>{});
    using CopyTiledShapeB = CopyTiledShapeA;
    using CopyThrdShapeB = CopyThrdShapeA;
    using SwizzleB = SwizzleA;
    using CopyTiledShapeC = decltype(neo::make_shape(neo::Int<CopyRowC>{}, neo::Int<CopyColC * CopyCount>{}));
    using CopyThrdShapeC = decltype(neo::make_shape(neo::Int<CopyRowC>{}, neo::Int<CopyColC>{}));

    static constexpr int PartitionA = 2;
    static constexpr int PartitionB = WrapCount / PartitionA;
    static constexpr int PartitionM = TILE_M / WrapCount * PartitionA;
    static constexpr int PartitionN = TILE_N / WrapCount * PartitionB;
    using TiledMmaShapeA = decltype(neo::make_shape(neo::Int<PartitionM>{}, neo::Int<MMA_K>{}));
    using TiledMmaShapeB = decltype(neo::make_shape(neo::Int<PartitionN>{}, neo::Int<MMA_K>{}));
    using TiledMmaShapeC = decltype(neo::make_shape(neo::Int<PartitionM>{}, neo::Int<PartitionN>{}));

    using MmaShapeA = decltype(neo::make_shape(neo::Int<MMA_M>{}, neo::Int<MMA_K>{}));
    using MmaShapeB = decltype(neo::make_shape(neo::Int<MMA_N>{}, neo::Int<MMA_K>{}));
    using MmaShapeC = decltype(neo::make_shape(neo::Int<MMA_N>{}, neo::Int<MMA_N>{}));
};

template <typename Config>
__global__ __launch_bounds__(128, 1)
void mma_aligned_128(Config::T* __restrict__ c, const Config::T* __restrict__ a, const Config::T* __restrict__ b, const int m, const int n, const int k) {
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;
    int wrapid = idx / Config::WrapSize;
    int laneid = idx % Config::WrapSize;
    int tiledM = wrapid / Config::PartitionA;
    int tiledN = wrapid % Config::PartitionB;

    using T = typename Config::T;
    extern __shared__ T shmem[];
    auto shm_a = shmem;
    auto shm_b = shmem + Config::ShmAsize;
    auto shm_c = shmem;

    using CopyTiledShapeA = typename Config::CopyTiledShapeA;
    using CopyThrdShapeA = typename Config::CopyThrdShapeA;
    using SwizzleA = typename Config::SwizzleA;
    auto copyTiledShapeA = CopyTiledShapeA{};
    auto copyThrdShapeA = CopyThrdShapeA{};
    auto copySwizzleA = SwizzleA{};

    auto A = neo::make_tensor(a, neo::make_shape(m, k), neo::make_stride(k, neo::Int<1>{}));
    auto gA = neo::local_tile(A, neo::make_shape(neo::Int<Config::TILE_M>{}, neo::Int<Config::TILE_K>{}), neo::make_coord(iy, neo::Int<0>{}));
    auto sA = neo::make_tensor(shm_a, neo::make_shape(neo::Int<Config::TILE_M>{}, neo::Int<Config::TILE_K>{}), neo::make_stride(neo::Int<Config::TILE_K>{}, neo::Int<1>{}));
    auto gOuterShapeA = neo::inner_div(gA.shape(), copyTiledShapeA);
    auto thrCopyCoordA = neo::copy_partition(copyThrdShapeA, idx, neo::Int<Config::CopyCount>{});

    using CopyTiledShapeB = typename Config::CopyTiledShapeB;
    using CopyThrdShapeB = typename Config::CopyThrdShapeB;
    using SwizzleB = typename Config::SwizzleB;
    auto copyTiledShapeB = CopyTiledShapeB{};
    auto copyThrdShapeB = CopyThrdShapeB{};
    auto copySwizzleB = SwizzleB{};

    auto B = neo::make_tensor(b, neo::make_shape(n, k), neo::make_stride(k, neo::Int<1>{}));
    auto gB = neo::local_tile(B, neo::make_shape(neo::Int<Config::TILE_N>{}, neo::Int<Config::TILE_K>{}), neo::make_coord(ix, neo::Int<0>{}));
    auto sB = neo::make_tensor(shm_b, neo::make_shape(neo::Int<Config::TILE_N>{}, neo::Int<Config::TILE_K>{}), neo::make_stride(neo::Int<Config::TILE_K>{}, neo::Int<1>{}));
    auto gOuterShapeB = neo::inner_div(gA.shape(), copyTiledShapeB);
    auto thrCopyCoordB = neo::copy_partition(copyThrdShapeB, idx, neo::Int<Config::CopyCount>{});

    using TiledMmaShapeA = typename Config::TiledMmaShapeA;
    auto tiledMmaShapeA = TiledMmaShapeA{};
    auto sOuterTiledMmaShapeA = neo::inner_div(sA.shape(), tiledMmaShapeA);
    auto tiledMmaA = neo::local_tile(sA, tiledMmaShapeA);

    using TiledMmaShapeB = typename Config::TiledMmaShapeB;
    auto tiledWrapMmaShapeB = TiledMmaShapeB{};
    auto sOuterTiledWrapMmaShapeB = neo::inner_div(sB.shape(), tiledWrapMmaShapeB);
    auto tiledMmaB = neo::local_tile(sB, tiledWrapMmaShapeB);

    using MmaShapeA = typename Config::MmaShapeA;
    auto mmaShapeA = MmaShapeA{};
    auto sOuterMmaShapeA = neo::inner_div(tiledMmaA.shape(), mmaShapeA);

    using MmaShapeB = typename Config::MmaShapeB;
    auto mmaShapeB = MmaShapeB{};
    auto sOuterMmaShapeB = neo::inner_div(tiledMmaB.shape(), mmaShapeB);

    auto ldmaCoordA = neo::make_coord(laneid % 16, (laneid / 16) * (16 / sizeof(T)));
    auto ldmaCoordB = ldmaCoordA;
    uint32_t a_frag[2][sOuterMmaShapeA.row_spacing][sOuterMmaShapeA.col_spacing][4];
    uint32_t b_frag[2][sOuterMmaShapeB.row_spacing][sOuterMmaShapeB.col_spacing][2][2];
    uint32_t c_frag[sOuterMmaShapeA.row_spacing][sOuterMmaShapeB.row_spacing][4] = {0};

    int itile_to_read = 0;
    int ismem_read = 0;
    int ismem_write = 0;

    // submit kStage - 1 tile
// gmem -> shm
#pragma unroll
    for (int istage = 0; istage < Config::Stage - 1; ++istage) {
        gA.jump(neo::make_coord(iy, istage));
        auto gAcopyTile = neo::local_tile(gA, copyTiledShapeA);
        auto sAcopyTile = neo::local_tile(sA, copyTiledShapeA);

        gB.jump(neo::make_coord(ix, istage));
        auto gBcopyTile = neo::local_tile(gB, copyTiledShapeB);
        auto sBcopyTile = neo::local_tile(sB, copyTiledShapeB);

        // copy global A to shared A
#pragma unroll
        for (int i = 0; i < gOuterShapeA.row_spacing; ++i) {
#pragma unroll
            for (int j = 0; j < gOuterShapeA.col_spacing; ++j) {
                auto coord = neo::make_coord(i, j);
                gAcopyTile.jump(coord);
                sAcopyTile.jump(coord);
                auto gAptr = gAcopyTile.move_at(thrCopyCoordA);
                auto sAptr = __cvta_generic_to_shared(sAcopyTile.base() + istage * (Config::ShmAsize + Config::ShmBsize) + copySwizzleA(sAcopyTile.crx2idx(thrCopyCoordA)));
                asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" :
                : "l"(sAptr), "l"(gAptr), "n"(Config::CopyUnitSize));
            }
        }

        // copy global B to shared B
#pragma unroll
        for (int i = 0; i < gOuterShapeB.row_spacing; ++i) {
#pragma unroll
            for (int j = 0; j < gOuterShapeB.col_spacing; ++j) {
                auto coord = neo::make_coord(i, j);
                gBcopyTile.jump(coord);
                sBcopyTile.jump(coord);
                auto gBptr = gBcopyTile.move_at(thrCopyCoordB);
                auto sBptr = __cvta_generic_to_shared(sBcopyTile.base() + istage * (Config::ShmAsize + Config::ShmBsize) + copySwizzleB(sBcopyTile.crx2idx(thrCopyCoordB)));
                asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" :
                : "l"(sBptr), "l"(gBptr), "n"(Config::CopyUnitSize));
            }
        }

        asm volatile("cp.async.commit_group;\n" ::);

        ++itile_to_read;
        ++ismem_write;
    }

    // wait one submitted gmem->smem done
    asm volatile("cp.async.wait_group %0;\n" :: "n"(Config::Stage - 2));
    __syncthreads();

    // smem -> reg
    {
        neo::Int<0> ik{};
        tiledMmaA.jump(neo::make_coord(tiledM, ik));
        tiledMmaB.jump(neo::make_coord(tiledN, ik));
        auto MmaA = neo::local_tile(tiledMmaA, mmaShapeA);
        auto MmaB = neo::local_tile(tiledMmaB, mmaShapeB);
#pragma unroll
        for (int mma_m = 0; mma_m < sOuterMmaShapeA.row_spacing; ++mma_m) {
#pragma unroll
            for (int mma_k = 0; mma_k < sOuterMmaShapeA.col_spacing; ++mma_k) {
                MmaA.jump(neo::make_coord(mma_m, mma_k));
                unsigned sAptr = __cvta_generic_to_shared(MmaA.base() + ismem_read * (Config::ShmAsize + Config::ShmBsize) + copySwizzleA(MmaA.crx2idx(ldmaCoordA)));
                asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 { %0, %1, %2, %3 }, [ %4 ];\n"
                    : "=r"(a_frag[ik][mma_m][mma_k][0]), "=r"(a_frag[ik][mma_m][mma_k][1]), "=r"(a_frag[ik][mma_m][mma_k][2]), "=r"(a_frag[ik][mma_m][mma_k][3])
                    : "r"(sAptr)
                    );
            }
        }

#pragma unroll
        for (int mma_n = 0; mma_n < sOuterMmaShapeB.row_spacing; ++mma_n) {
#pragma unroll
            for (int mma_k = 0; mma_k < sOuterMmaShapeB.col_spacing; ++mma_k) {
                MmaB.jump(neo::make_coord(mma_n, mma_k));
                unsigned sBptr = __cvta_generic_to_shared(MmaB.base() + ismem_read * (Config::ShmAsize + Config::ShmBsize) + copySwizzleB(MmaB.crx2idx(ldmaCoordB)));
                asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 { %0, %1, %2, %3 }, [ %4 ];\n"
                    : "=r"(b_frag[ik][mma_n][mma_k][0][0]), "=r"(b_frag[ik][mma_n][mma_k][1][0]), "=r"(b_frag[ik][mma_n][mma_k][0][1]), "=r"(b_frag[ik][mma_n][mma_k][1][1])
                    : "r"(sBptr)
                    );
            }
        }
    }

#pragma unroll
    for (int itile = 0, ntile = k / Config::TILE_K; itile < ntile; ++itile) {
#pragma unroll
        for (int ik = 0, nk = sOuterTiledMmaShapeA.col_spacing; ik < nk; ++ik) {
            int ik_next = (ik + 1) % nk;

            if (ik == nk - 1) {
                asm volatile("cp.async.wait_group %0;\n" :: "n"(Config::Stage - 2));
                __syncthreads();

                ismem_read = (ismem_read + 1) % Config::Stage;
            }

            tiledMmaA.jump(neo::make_coord(tiledM, ik_next));
            tiledMmaB.jump(neo::make_coord(tiledN, ik_next));
            auto MmaA = neo::local_tile(tiledMmaA, mmaShapeA);
            auto MmaB = neo::local_tile(tiledMmaB, mmaShapeB);

#pragma unroll
            for (int mma_m = 0; mma_m < sOuterMmaShapeA.row_spacing; ++mma_m) {
#pragma unroll
                for (int mma_k = 0; mma_k < sOuterMmaShapeA.col_spacing; ++mma_k) {
                    MmaA.jump(neo::make_coord(mma_m, mma_k));
                    unsigned sAptr = __cvta_generic_to_shared(MmaA.base() + ismem_read * (Config::ShmAsize + Config::ShmBsize) + copySwizzleA(MmaA.crx2idx(ldmaCoordA)));
                    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 { %0, %1, %2, %3 }, [ %4 ];\n"
                        : "=r"(a_frag[ik_next][mma_m][mma_k][0]), "=r"(a_frag[ik_next][mma_m][mma_k][1]), "=r"(a_frag[ik_next][mma_m][mma_k][2]), "=r"(a_frag[ik_next][mma_m][mma_k][3])
                        : "r"(sAptr)
                    );
                }
            }

#pragma unroll
            for (int mma_n = 0; mma_n < sOuterMmaShapeB.row_spacing; ++mma_n) {
#pragma unroll
                for (int mma_k = 0; mma_k < sOuterMmaShapeB.col_spacing; ++mma_k) {
                    MmaB.jump(neo::make_coord(mma_n, mma_k));
                    unsigned sBptr = __cvta_generic_to_shared(MmaB.base() + ismem_read * (Config::ShmAsize + Config::ShmBsize) + copySwizzleB(MmaB.crx2idx(ldmaCoordB)));
                    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 { %0, %1, %2, %3 }, [ %4 ];\n"
                        : "=r"(b_frag[ik_next][mma_n][mma_k][0][0]), "=r"(b_frag[ik_next][mma_n][mma_k][1][0]), "=r"(b_frag[ik_next][mma_n][mma_k][0][1]), "=r"(b_frag[ik_next][mma_n][mma_k][1][1])
                        : "r"(sBptr)
                    );
                }
            }

            if (ik == 0) {
                if (itile_to_read < ntile) {
                    gA.jump(neo::make_coord(iy, itile_to_read));
                    auto gAcopyTile = neo::local_tile(gA, copyTiledShapeA);
                    auto sAcopyTile = neo::local_tile(sA, copyTiledShapeA);

                    gB.jump(neo::make_coord(ix, itile_to_read));
                    auto gBcopyTile = neo::local_tile(gB, copyTiledShapeB);
                    auto sBcopyTile = neo::local_tile(sB, copyTiledShapeB);

                    // copy global A to shared A
#pragma unroll
                    for (int i = 0; i < gOuterShapeA.row_spacing; ++i) {
#pragma unroll
                        for (int j = 0; j < gOuterShapeA.col_spacing; ++j) {
                            auto coord = neo::make_coord(i, j);
                            gAcopyTile.jump(coord);
                            sAcopyTile.jump(coord);
                            auto gAptr = gAcopyTile.move_at(thrCopyCoordA);
                            auto sAptr = __cvta_generic_to_shared(sAcopyTile.base() + ismem_write * (Config::ShmAsize + Config::ShmBsize) + copySwizzleA(sAcopyTile.crx2idx(thrCopyCoordA)));
                            asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" :
                            : "l"(sAptr), "l"(gAptr), "n"(Config::CopyUnitSize));
                        }
                    }

                    // copy global B to shared B
#pragma unroll
                    for (int i = 0; i < gOuterShapeB.row_spacing; ++i) {
#pragma unroll
                        for (int j = 0; j < gOuterShapeB.col_spacing; ++j) {
                            auto coord = neo::make_coord(i, j);
                            gBcopyTile.jump(coord);
                            sBcopyTile.jump(coord);
                            auto gBptr = gBcopyTile.move_at(thrCopyCoordB);
                            auto sBptr = __cvta_generic_to_shared(sBcopyTile.base() + ismem_write * (Config::ShmAsize + Config::ShmBsize) + copySwizzleB(sBcopyTile.crx2idx(thrCopyCoordB)));
                            asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" :
                            : "l"(sBptr), "l"(gBptr), "n"(Config::CopyUnitSize));
                        }
                    }

                    ++itile_to_read;
                    ismem_write = (ismem_write + 1) % Config::Stage;
                }

                asm volatile("cp.async.commit_group;\n" ::);
            }

#pragma unroll
            for (int mma_m = 0; mma_m < sOuterMmaShapeA.row_spacing; ++mma_m) {
#pragma unroll
                for (int mma_n = 0; mma_n < sOuterMmaShapeB.row_spacing; ++mma_n) {
#pragma unroll
                    for (int mma_k = 0; mma_k < sOuterMmaShapeB.col_spacing; ++mma_k) {
                        asm volatile(
                            "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 "
                            "{%0,  %1},"
                            "{%2,  %3,  %4,  %5},"
                            "{%6,  %7},"
                            "{%8,  %9};\n"
                            : "=r"(c_frag[mma_m][mma_n][0]), "=r"(c_frag[mma_m][mma_n][1])
                            : "r"(a_frag[ik][mma_m][mma_k][0]), "r"(a_frag[ik][mma_m][mma_k][1]), "r"(a_frag[ik][mma_m][mma_k][2]), "r"(a_frag[ik][mma_m][mma_k][3]),
                            "r"(b_frag[ik][mma_n][mma_k][0][0]), "r"(b_frag[ik][mma_n][mma_k][0][1]),
                            "r"(c_frag[mma_m][mma_n][0]), "r"(c_frag[mma_m][mma_n][1]));

                        asm volatile(
                            "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 "
                            "{%0,  %1},"
                            "{%2,  %3,  %4,  %5},"
                            "{%6,  %7},"
                            "{%8,  %9};\n"
                            : "=r"(c_frag[mma_m][mma_n][2]), "=r"(c_frag[mma_m][mma_n][3])
                            : "r"(a_frag[ik][mma_m][mma_k][0]), "r"(a_frag[ik][mma_m][mma_k][1]), "r"(a_frag[ik][mma_m][mma_k][2]), "r"(a_frag[ik][mma_m][mma_k][3]),
                            "r"(b_frag[ik][mma_n][mma_k][1][0]), "r"(b_frag[ik][mma_n][mma_k][1][1]),
                            "r"(c_frag[mma_m][mma_n][2]), "r"(c_frag[mma_m][mma_n][3]));
                    }
                }
            }
        }

        __syncthreads();
    }

    auto sC = neo::make_tensor(shm_c, neo::make_shape(neo::Int<Config::TILE_M>{}, neo::Int<Config::TILE_N>{}), neo::make_stride(neo::Int<Config::TILE_N>{}, neo::Int<1>{}));
    using TiledMmaShapeC = typename Config::TiledMmaShapeC;
    auto tiledWrapMmaShapeC = TiledMmaShapeC{};
    auto tiledMmaC = neo::local_tile(sC, tiledWrapMmaShapeC, neo::make_coord(tiledM, tiledN));

    using MmaShapeC = typename Config::MmaShapeC;
    auto mmaShapeC = MmaShapeC{};
    auto sOuterMmaShapeC = neo::inner_div(tiledMmaC.shape(), mmaShapeC);
    auto MmaC = neo::local_tile(tiledMmaC, mmaShapeC);

    using CopyTiledShapeC = typename Config::CopyTiledShapeC;
    using CopyThrdShapeC = typename Config::CopyThrdShapeC;
    auto copyTiledShapeC = CopyTiledShapeC{};
    auto copyThrdShapeC = CopyThrdShapeC{};

    auto C = neo::make_tensor(c, neo::make_shape(m, n), neo::make_stride(n, neo::Int<1>{}));
    auto gC = neo::local_tile(C, neo::make_shape(neo::Int<Config::TILE_M>{}, neo::Int<Config::TILE_N>{}), neo::make_coord(iy, ix));
    auto gOuterShapeC = neo::inner_div(gC.shape(), copyTiledShapeC);
    auto thrCopyCoordC = neo::copy_partition(copyThrdShapeC, idx, neo::Int<Config::CopyCount>{});
    auto gCcopyTile = neo::local_tile(gC, copyTiledShapeC);
    auto sCcopyTile = neo::local_tile(sC, copyTiledShapeC);

    int c_row0 = laneid / 4;
    int c_row1 = laneid / 4 + 16 / sizeof(T);
    int c_col0 = laneid % 4 * sizeof(uint32_t) / sizeof(Config::T);
    int c_col1 = laneid % 4 * sizeof(uint32_t) / sizeof(Config::T) + 16 / sizeof(T);

    // reg C to shared C
#pragma unroll
    for (int im = 0; im < sOuterMmaShapeC.row_spacing; ++im) {
#pragma unroll
        for (int in = 0; in < sOuterMmaShapeC.col_spacing; ++in) {
            auto cPtr = MmaC.jump_at(neo::make_coord(im, in));
            *(uint32_t*)(&cPtr[neo::dot(neo::make_coord(c_row0, c_col0), MmaC.stride())]) = c_frag[im][in][0];
            *(uint32_t*)(&cPtr[neo::dot(neo::make_coord(c_row1, c_col0), MmaC.stride())]) = c_frag[im][in][1];
            *(uint32_t*)(&cPtr[neo::dot(neo::make_coord(c_row0, c_col1), MmaC.stride())]) = c_frag[im][in][2];
            *(uint32_t*)(&cPtr[neo::dot(neo::make_coord(c_row1, c_col1), MmaC.stride())]) = c_frag[im][in][3];
        }
    }

    __syncthreads();

    // copy shared C to global C
#pragma unroll
    for (int i = 0; i < gOuterShapeC.row_spacing; ++i) {
#pragma unroll
        for (int j = 0; j < gOuterShapeC.col_spacing; ++j) {
            auto coord = neo::make_coord(i, j);
            gCcopyTile.jump(coord);
            sCcopyTile.jump(coord);
            auto gCptr = reinterpret_cast<int4*>(gCcopyTile.move_at(thrCopyCoordC));
            auto sCptr = reinterpret_cast<int4*>(sCcopyTile.move_at(thrCopyCoordC));
            *gCptr = *sCptr;
        }
    }
}

int main() {
    int m = NeoConfig::TILE_M * 640, n = NeoConfig::TILE_N * 2, k = NeoConfig::TILE_K * 16;

    thrust::host_vector<NeoConfig::T> h_a(m * k);
    thrust::host_vector<NeoConfig::T> h_b(n * k);

    std::mt19937 gen(10086);
    std::uniform_int_distribution<> dis(0, 200);

    for (auto& i : h_a) {
        float nn = ((dis(gen) % 200) - 100.0f) * 0.01f;
        i = nn;
    }

    for (auto& i : h_b) {
        float nn = ((dis(gen) % 200) - 100.0f) * 0.01f;
        i = nn;
    }

    thrust::device_vector<NeoConfig::T> d_a = h_a;
    thrust::device_vector<NeoConfig::T> d_b = h_b;
    thrust::device_vector<NeoConfig::T> d_c1(m * n, 0);
    thrust::device_vector<NeoConfig::T> d_c2(m * n, 0);

    dim3 grid(n / NeoConfig::TILE_N, m / NeoConfig::TILE_M);
    dim3 block(NeoConfig::Threads);

    // cuda warmup
    constexpr int warmup = 100;
    constexpr int nt = 1000;
    hipFuncSetAttribute(reinterpret_cast<const void*>(mma_aligned_128<NeoConfig>),
        hipFuncAttributeMaxDynamicSharedMemorySize, NeoConfig::ShmSize);

    for (int i = 0; i < warmup; ++i) {
        mma_aligned_128<NeoConfig> << <grid, block, NeoConfig::ShmSize >> > (d_c1.data().get(), d_a.data().get(), d_b.data().get(), m, n, k);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_mma = 0;

    for (int i = 0; i < nt; ++i) {
        hipEventRecord(start);

        mma_aligned_128<NeoConfig> << <grid, block, NeoConfig::ShmSize >> > (d_c1.data().get(), d_a.data().get(), d_b.data().get(), m, n, k);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed = 0;
        hipEventElapsedTime(&elapsed, start, stop);
        time_mma += elapsed;

        auto err = hipGetLastError();
        if (err != hipSuccess) {
            std::cout << "mma_aligned_128 launch error: " << err << std::endl;
        }
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    NeoConfig::T alpha = NeoConfig::T(1.f);
    NeoConfig::T beta = NeoConfig::T(0.f);

    // cublas warmup
    for (int i = 0; i < warmup; ++i) {
        hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            n, m, k,
            &alpha,
            d_b.data().get(), k,
            d_a.data().get(), k,
            &beta,
            d_c2.data().get(), n);
    }

    float time_cublas = 0;

    for (int i = 0; i < nt; ++i) {
        hipEventRecord(start);

        hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            n, m, k,
            &alpha,
            d_b.data().get(), k,
            d_a.data().get(), k,
            &beta,
            d_c2.data().get(), n);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed = 0;
        hipEventElapsedTime(&elapsed, start, stop);
        time_cublas += elapsed;

        if (ret != hipSuccess) {
            std::cout << "hipblasHgemm error: " << ret << std::endl;
        }
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    gpu_compare(d_c1.data().get(), d_c2.data().get(), m * n);

    std::cout << "my mma: " << time_mma / nt << " cublas: " << time_cublas / nt << std::endl;

    return 0;
}
